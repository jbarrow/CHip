#include <stdio.h>
#include <time.h>

extern "C" {
	#include "../src/logger.h"
	#include "../src/linear/matrix.h"
}

#define NUM_MATRICES 3

static int timestamp = 0;
static char filename[20];

char* get_filename() {
	if(timestamp == 0) {
		timestamp = (int)time(NULL);
		sprintf(filename, "benchmark_%d.csv", timestamp);
	}

	return filename;
}

/*
 *	Initialize the benchmark variable, which will propogate
 *	through the implementation code.
 */
int main() {
	/* Initialize matrix values */
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float elapsed_time;
	char content[200];
	c_matrix* matrices[3*NUM_MATRICES];

	/* Loop through matrix sizes and create matrices */
	int i;
	unsigned int j = 1;
	for(i = 0; i < NUM_MATRICES; ++i) {
		matrices[3*i] = new_c_matrix(j, j);
		matrices[3*i+1] = new_c_matrix(j, j);
		matrices[3*i+2] = new_c_matrix(j, j);
		j = j << 1;
	}
	
	/* Create the CSV header for our benchmark spreadsheet */
	sprintf(content, "Operation, Matrix Size, Time Taken (ms)\n");
	push_to_log_file(content, get_filename());

	/* Matrix Initialization */
	j = 1;
	for(i = 0; i < NUM_MATRICES; ++i) {
		hipEventRecord(start, 0);

		ones(matrices[3*i]);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time, start, stop);
		sprintf(content, "Initialize to 1, %d, %f\n", j, elapsed_time);
		push_to_log_file(content, get_filename());
		j = j << 1;
	}

	/* Matrix Initialization */
	j = 1;
	for(i = 0; i < NUM_MATRICES; ++i) {
		hipEventRecord(start, 0);

		zeros(matrices[3*i+1]);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time, start, stop);
		sprintf(content, "Initialize to 0, %d, %f\n", j, elapsed_time);
		push_to_log_file(content, get_filename());
		j = j << 1;
	}

	/* Matrix Addition */
	j = 1;
	for(i = 0; i < NUM_MATRICES; ++i) {
		hipEventRecord(start, 0);

		c_matrix_add(matrices[3*i], matrices[3*i+1], matrices[3*i+2]);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time, start, stop);
		sprintf(content, "Add Two Matrices, %d, %f\n", j, elapsed_time);
		push_to_log_file(content, get_filename());
		j = j << 1;
	}
	

	return(EXIT_SUCCESS);
}