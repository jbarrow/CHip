#include "hip/hip_runtime.h"
#include "matrix_functions.cu"
#include <stdlib.h>

extern "C" {
	#include "matrix.h"
}

__global__ void cu_matrix_add(const double *d_a, const double *d_b, double *d_c, int element_count);
static void handleError( hipError_t err, const char *file, int line );

extern "C"
c_matrix *new_c_matrix(int i, int j) {
	c_matrix *m = (c_matrix *)malloc(sizeof(*m));
	if(m == NULL)
		return NULL;

	m->data = (double *)malloc(sizeof(double) * i * j);
	if(m->data == NULL) {
		free(m);
		return NULL;
	}

	m->n = i;
	m->m = j;

	return m;
}

extern "C"
void del_c_matrix(c_matrix *m) {
	if( m != NULL ) {
		free(m->data);
		free(m);
	}
}

extern "C"
double c_matrix_get(c_matrix *m, int i, int j) {
	/* 	Ideally if we reach this point, we could raise an error;
	 *	Still trying to figure out how to handle errors in C */
	if(i > m->n || j > m->m || i < 1 || j < 1)
		return NULL;
	return m->data[(i-1) * (j-1) + (j-1)];
}

/*
 *	Set a specific element to a specific value, with index
 *	checking.
 */
extern "C"
void c_matrix_set(c_matrix *m, int i, int j, double value) {
	if(i > m->n || j > m->m || i < 1 || j < 1)
		exit(EXIT_FAILURE);
	m->data[(i-1) * (j-1) + (i-1)] = value;
}

/*	Initialize a matrix with all zero's */
extern "C"
void zeros(c_matrix *m) {
	/* memcpy(m->data, 0, sizeof(double) * m->m * m->n); */
	int i;
	for(i = 0; i < m->m * m->n; ++i)
		m->data[i] = 0.0;
}

/*	Initialize a matrix with all one's */
extern "C"
void ones(c_matrix *m) {
	int i;
	for(i = 0; i < m->m * m->n; ++i)
		m->data[i] = 1.0;
}

/*
 *	Initializes the matrix with random numbers defined
 *	over the interval (a, b]
 */
extern "C"
void c_matrix_rand(c_matrix *m1, double a, double b) {}

/*
 *	Takes in a function pointer representing a specific
 *	distribution that returns a random double. In neural
 *	networks, it is sometimes favorable to intitialize
 *	a matrix with a t-distribution
 */
extern "C"
void c_matrix_dist(c_matrix *m1, double (*init)()) {}

/*	Add two matrices and store the result in a third */
extern "C"
void c_matrix_add(const c_matrix *m1, const c_matrix *m2, c_matrix *m) {
	/*	We only need 4 comparisons because we can assume
	 *	transitivity of ints */
	if(m1->m != m2->m || m1->n != m2->n || m1->m != m->m
		|| m1->n != m2->n)
		exit(EXIT_FAILURE);

	double *d_a, *d_b, *d_c;
	handle_error( hipMalloc((void**)&d_a, m1->m * m1->n * sizeof(double)) );
	handle_error( hipMalloc((void**)&d_b, m1->m * m1->n * sizeof(double)) );
	handle_error( hipMalloc((void**)&d_c, m1->m * m1->n * sizeof(double)) );

	handle_error( hipMemcpy(d_a, m1->data, m1->m * m1->n * sizeof(double), hipMemcpyHostToDevice ) );
	handle_error( hipMemcpy(d_b, m2->data, m2->m * m2->n * sizeof(double), hipMemcpyHostToDevice ) );

	cu_matrix_add<<< 32, (m1->m * m1->n + 31)/32 >>>(d_a, d_b, d_c, m1->m * m1->n);

	handle_error( hipMemcpy(m->data, d_c, m2->m * m2->n * sizeof(double), hipMemcpyDeviceToHost ) );

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
}

/*	Subtract two matrices and store the result in a third */
extern "C"
void c_matrix_sub(const c_matrix *m1, const c_matrix *m2, c_matrix *m) {
	if(m1->m != m2->m || m1->n != m2->n || m1->m != m->m
		|| m1->n != m2->n)
		exit(EXIT_FAILURE);

	double *d_a, *d_b, *d_c;
	handle_error( hipMalloc((void**)&d_a, m1->m * m1->n * sizeof(double)) );
	handle_error( hipMalloc((void**)&d_b, m1->m * m1->n * sizeof(double)) );
	handle_error( hipMalloc((void**)&d_c, m1->m * m1->n * sizeof(double)) );

	handle_error( hipMemcpy(d_a, m1->data, m1->m * m1->n * sizeof(double), hipMemcpyHostToDevice ) );
	handle_error( hipMemcpy(d_b, m2->data, m2->m * m2->n * sizeof(double), hipMemcpyHostToDevice ) );

	cu_matrix_sub<<< 32, (m1->m * m1->n + 31)/32 >>>(d_a, d_b, d_c, m1->m * m1->n);

	handle_error( hipMemcpy(m->data, d_c, m2->m * m2->n * sizeof(double), hipMemcpyDeviceToHost ) );

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
}

/*	Element-wise multiply two matrices and store the result in a third */
extern "C"
void c_element_mul(const c_matrix *m1, const c_matrix *m2, c_matrix *m) {
	if(m1->m != m2->m || m1->n != m2->n || m1->m != m->m
		|| m1->n != m2->n)
		exit(EXIT_FAILURE);

	double *d_a, *d_b, *d_c;
	handle_error( hipMalloc((void**)&d_a, m1->m * m1->n * sizeof(double)) );
	handle_error( hipMalloc((void**)&d_b, m1->m * m1->n * sizeof(double)) );
	handle_error( hipMalloc((void**)&d_c, m1->m * m1->n * sizeof(double)) );

	handle_error( hipMemcpy(d_a, m1->data, m1->m * m1->n * sizeof(double), hipMemcpyHostToDevice ) );
	handle_error( hipMemcpy(d_b, m2->data, m2->m * m2->n * sizeof(double), hipMemcpyHostToDevice ) );

	cu_element_mul<<< 32, (m1->m * m1->n + 31)/32 >>>(d_a, d_b, d_c, m1->m * m1->n);

	handle_error( hipMemcpy(m->data, d_c, m2->m * m2->n * sizeof(double), hipMemcpyDeviceToHost ) );

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
}

/*	Element-wise divide two matrices and store the result in a third */
extern "C"
void c_element_div(const c_matrix *m1, const c_matrix *m2, c_matrix *m) {
	if(m1->m != m2->m || m1->n != m2->n || m1->m != m->m
		|| m1->n != m2->n)
		exit(EXIT_FAILURE);

	double *d_a, *d_b, *d_c;
	handle_error( hipMalloc((void**)&d_a, m1->m * m1->n * sizeof(double)) );
	handle_error( hipMalloc((void**)&d_b, m1->m * m1->n * sizeof(double)) );
	handle_error( hipMalloc((void**)&d_c, m1->m * m1->n * sizeof(double)) );

	handle_error( hipMemcpy(d_a, m1->data, m1->m * m1->n * sizeof(double), hipMemcpyHostToDevice ) );
	handle_error( hipMemcpy(d_b, m2->data, m2->m * m2->n * sizeof(double), hipMemcpyHostToDevice ) );

	cu_element_div<<< 32, (m1->m * m1->n + 31)/32 >>>(d_a, d_b, d_c, m1->m * m1->n);

	handle_error( hipMemcpy(m->data, d_c, m2->m * m2->n * sizeof(double), hipMemcpyDeviceToHost ) );

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
}