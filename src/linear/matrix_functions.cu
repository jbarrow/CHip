#include "hip/hip_runtime.h"
#include <stdio.h>

/*
 *  Code provided by NVidia to be able to handle atomic adds
 *  for double-precision floating point numbers
 */
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void cu_matrix_add(const double *d_a, const double *d_b, double *d_c, int element_count) {
    
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = x * y + x;

    while( tid < element_count ) {
        d_c[tid] = d_a[tid] + d_b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void cu_matrix_sub(const double *d_a, const double *d_b, double *d_c, int element_count) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = x * y + x;

    while( tid < element_count ) {
        d_c[tid] = d_a[tid] - d_b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void cu_element_mul(const double *d_a, const double *d_b, double *d_c, int element_count) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = x * y + x;

    while( tid < element_count ) {
        d_c[tid] = d_a[tid] * d_b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void cu_element_div(const double *d_a, const double *d_b, double *d_c, int element_count) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = x * y + x;

    while( tid < element_count ) {
        d_c[tid] = d_a[tid] / d_b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

static void handleError( hipError_t err, const char* file, int line ) {
    if(err != hipSuccess) {
        printf("%s (%d): %s", file, line, hipGetErrorString(err));
        exit( EXIT_FAILURE );
    }
}